#include "hip/hip_runtime.h"
#include "helpers.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>

using namespace std;

#define DTYPE float
//#define M 3 
//#define N 2
#define M 512
#define N 512
#define K 512

// Error checking macro for CUDA Runtime API
#define cudaCheck(error) \
    do { \
        hipError_t err = (error); \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)


// Variables
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;
hipFunction_t sgemm_kernel;
DTYPE *h_A, *h_B, *h_C, *h_C_ref;
hipDeviceptr_t d_A, d_B, d_C, d_C_ref;
hipDeviceptr_t d_A_col, d_B_col;
hipblasHandle_t handle;

// Function to run cuBLAS kernel
void run_cublas_kernel(float alpha, hipDeviceptr_t dA, hipDeviceptr_t dB, float beta, hipDeviceptr_t dC) {
    float* dA_ptr = reinterpret_cast<float*>(dA);
    float* dB_ptr = reinterpret_cast<float*>(dB);
    float* dC_ptr = reinterpret_cast<float*>(dC);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dB_ptr, HIP_R_32F,
               N, dA_ptr, HIP_R_32F, K, &beta, dC_ptr, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

// Function to verify matrix results
bool verify_matrix(DTYPE* ref, DTYPE* test, int size, float epsilon = 1e-2) {
    for (int i = 0; i < size; i++) {
        if (fabs(ref[i] - test[i]) > epsilon) {
            std::cout << "Mismatch at index " << i << ": ref=" << ref[i] << ", test=" << test[i] << std::endl;
            return false;
        }
    }
    return true;
}

// Function to print matrix
void print_matrix(DTYPE* matrix, int rows, int cols, std::ostream& os = std::cout) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            os << matrix[i * cols + j] << " ";
        }
        os << std::endl;
    }
}

int main(int argc, char **argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <kernel_name> <module_path>\n";
        return 1;
    }
    std::string kernel_name = argv[1];
    std::string module_path = argv[2];
    int devID = 0;

    // Initialize cuBLAS
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };

    // Create events for timing

    // Initialize CUDA
    checkCudaErrors(hipInit(0));
    checkCudaErrors(hipDeviceGet(&cuDevice, devID));
    checkCudaErrors(hipCtxCreate(&cuContext, 0, cuDevice));
    checkCudaErrors(hipModuleLoad(&cuModule, module_path.c_str()));
    checkCudaErrors(hipModuleGetFunction(&sgemm_kernel, cuModule, kernel_name.c_str()));

    // Matrix dimensions and sizes
    int sizeA = M * K * sizeof(DTYPE);
    int sizeB = K * N * sizeof(DTYPE);
    int sizeC = M * N * sizeof(DTYPE);
    
    // Allocate host memory
    h_A = (DTYPE *)malloc(sizeA);
    h_B = (DTYPE *)malloc(sizeB);
    h_C = (DTYPE *)malloc(sizeC);
    h_C_ref = (DTYPE *)malloc(sizeC);

    // Initialize matrices
    initMatrix(h_A, M*K, InitMode::RANDOM);
    initMatrix(h_B, K*N, InitMode::RANDOM);


    // Allocate device memory
    checkCudaErrors(hipMalloc(&d_A, sizeA));
    checkCudaErrors(hipMalloc(&d_B, sizeB));
    checkCudaErrors(hipMalloc(&d_C, sizeC));
    checkCudaErrors(hipMalloc(&d_C_ref, sizeC));
    
    // Copy data to device
    checkCudaErrors(hipMemcpyHtoD(d_A, h_A, sizeA));
    checkCudaErrors(hipMemcpyHtoD(d_B, h_B, sizeB));
    
    // Initialize output matrices to zero
    checkCudaErrors(hipMemsetD8(d_C, 0, sizeC));
    checkCudaErrors(hipMemsetD8(d_C_ref, 0, sizeC));

    // Parameters for GEMM
    float alpha = 1.0f;
    float beta = 0.0f;

    // Run cuBLAS kernel for reference
    run_cublas_kernel(alpha, d_A, d_B, beta, d_C_ref);
    cudaCheck(hipDeviceSynchronize());
    
    // Run custom kernel
    void *args[] = { &d_C, &d_A, &d_B };
    checkCudaErrors(
        hipModuleLaunchKernel(sgemm_kernel, 
            //2, 3, 1,    // gridDim x, y, z
            512, 512, 1,    // gridDim x, y, z
            1, 1, 1,    // blockDim x, y, z
            0,           // sharedMemBytes
            NULL,        // hStream
            args,        // kernel params
            NULL         // extra
        )
    );
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipGetLastError());

    // Copy results back to host
    checkCudaErrors(hipMemcpyDtoH(h_C, d_C, sizeC));
    checkCudaErrors(hipMemcpyDtoH(h_C_ref, d_C_ref, sizeC));

    // Verify results
    printf("M %d N %d K %d\n", M, N, K);
    if (!verify_matrix(h_C_ref, h_C, M * N)) {
        std::cout << "Failed to pass the correctness verification against NVIDIA cuBLAS." << std::endl;
        
        // For small matrices, print the details
        if (M <= 128) {
            std::cout << "A:" << std::endl;
            print_matrix(h_A, M, K);
            std::cout << "B:" << std::endl;
            print_matrix(h_B, K, N);
            std::cout << "Custom kernel result:" << std::endl;
            print_matrix(h_C, M, N);
            std::cout << "cuBLAS result:" << std::endl;
            print_matrix(h_C_ref, M, N);
        }
        exit(EXIT_FAILURE);
    } else {
        std::cout << "Results match cuBLAS reference!" << std::endl;
    }

    // Time custom kernel
    float elapsed_time_custom;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    int repeat_times = 50;
    hipEventRecord(beg);
    for (int j = 0; j < repeat_times; j++) {
        checkCudaErrors(
            hipModuleLaunchKernel(sgemm_kernel, 
                N, M, 1,    // gridDim x, y, z
                1, 1, 1,    // blockDim x, y, z
                0,           // sharedMemBytes
                NULL,        // hStream
                args,        // kernel params
                NULL         // extra
            )
        );
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&elapsed_time_custom, beg, end);
    elapsed_time_custom /= 1000.0f; // Convert to seconds

    // Time cuBLAS kernel
    hipEventRecord(beg);
    for (int j = 0; j < repeat_times; j++) {
        run_cublas_kernel(alpha, d_A, d_B, beta, d_C_ref);
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    
    float elapsed_time_cublas;
    hipEventElapsedTime(&elapsed_time_cublas, beg, end);
    elapsed_time_cublas /= 1000.0f; // Convert to seconds

    // Calculate performance
    long flops = 2 * M * N * K;
    printf("Custom kernel - Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS\n",
           elapsed_time_custom / repeat_times,
           (repeat_times * flops * 1e-9) / elapsed_time_custom);
    printf("cuBLAS kernel - Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS\n",
           elapsed_time_cublas / repeat_times,
           (repeat_times * flops * 1e-9) / elapsed_time_cublas);

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_C_ref));
    hipblasDestroy(handle);
    hipEventDestroy(beg);
    hipEventDestroy(end);

    return 0;
}
