#include "helpers.h"

using namespace std;

// Variables
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;
hipFunction_t sgemm_kernel;
float *h_A;
hipDeviceptr_t d_A;

int main(int argc, char **argv)
{
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <kernel_name> <module_path>\n";
        return 1;
    }
    std::string kernel_name = argv[1];
    std::string module_path = argv[2];
    int devID = 0;

    checkCudaErrors(hipInit(0));
    checkCudaErrors(hipDeviceGet(&cuDevice, devID));
    checkCudaErrors(hipCtxCreate(&cuContext, 0, cuDevice));
    checkCudaErrors(hipModuleLoad(&cuModule, module_path.c_str()));
    checkCudaErrors(hipModuleGetFunction(&sgemm_kernel, cuModule, kernel_name.c_str()));

    int M = 3;
    int sizeA = M * sizeof(float);
    h_A = (float *)malloc(sizeA);
    checkCudaErrors(hipMalloc(&d_A, sizeA));

    void *args[] = { &d_A };
    checkCudaErrors(
        hipModuleLaunchKernel(sgemm_kernel, 
            1, 1, 1, // blockIdx x, y, z
            3, 1, 1, // threadIdx x, y, z
            0, // Shared mem bytes
            NULL, // hStream
            args, // Kernel params
            NULL // extra
        )
    );
    checkCudaErrors(hipMemcpyDtoH(h_A, d_A, sizeA));
    for (int i=0; i < M; i++) {
        printf("%f\n", h_A[i]);
    }
}
